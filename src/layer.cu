#include "hip/hip_runtime.h"
#include "layer.h"


/* Conv1D 
 * @param [in1]  in: [C, s]
 * @param [in2]   w: [OC, C, K] 
 * @param [in3]   b: [OC]
 * @param [out] out: [OC, os]
 *    
 *    In this model, K is 3, 5, 7, or 9, 
 *    with stride = 1, pad = 0, dilation = 1.
 *    The formula for the output sequence length:
 *      os = (in - K + 2 * pad) / stride + 1
 *         = (s - K + 2 * 0) / 1 + 1
 *         = s - K + 1
 *
 * 'C' is the input channel size
 * 's' is the input sequence length
 * 'OC' is the output channel size
 * 'os' is the output sequence length
 * 'K' is the kernel (or filter) size
 */
void Conv1D(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
  size_t s = in->shape[1];
  size_t C = in->shape[0];
  size_t OC = w->shape[0];
  size_t K = w->shape[2];
  
  size_t os = s - K + 1;

  for (size_t i = 0; i < OC; i++) {
    for (size_t j = 0; j < os; j++) {
      float val = 0.f;
      for (size_t k = 0; k < C; k++) {
        for (size_t l = 0; l < K; l++) {
          val += in->buf[k * s + j + l] * 
                  w->buf[i * C * K + k * K + l];
        }
      }
      out->buf[i * os + j] = val + b->buf[i];
    }
  }
}

/* ReLU
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
void ReLU(Tensor *inout) {
  size_t N = inout->num_elem();

  for (size_t i = 0; i < N; i++) {
    inout->buf[i] = inout->buf[i] > 0 ? inout->buf[i] : 0;
  }
}
/* [Example] ReLU CUDA kernel */
__global__ void ReLU_Kernel(float *inout, size_t N) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    inout[i] = inout[i] > 0 ? inout[i] : 0;
  }
}
/* [Example] ReLU using CUDA */
void ReLU_CUDA(Tensor *inout) {
  size_t N = inout->num_elem();

  float *d_inout;
  CHECK_CUDA(hipMalloc(&d_inout, N * sizeof(float)));
  CHECK_CUDA(hipMemcpy(d_inout, inout->buf, N * sizeof(float), 
                        hipMemcpyHostToDevice));

  ReLU_Kernel<<<(N + 255) / 256, 256>>>(d_inout, N);
  CHECK_CUDA(hipDeviceSynchronize());

  CHECK_CUDA(hipMemcpy(inout->buf, d_inout, N * sizeof(float), 
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipFree(d_inout));
}

/* GetMax
 * @param [in]   in: [C, s]
 * @param [out] out: [C]
 *    
 *    This layer is to get the max value along the sequence dim.
 *    The formula for this layer: out = max(in, dim=-1)
 * 
 * 'C' is the channel size
 * 's' is the sequence length
 */
void GetMax(Tensor *in, Tensor *out) {
  size_t C = in->shape[0];
  size_t s = in->shape[1];

  for (size_t i = 0; i < C; i++) {
    out->buf[i] = in->buf[i * s];
    for (size_t j = 1; j < s; j++) {
      out->buf[i] = in->buf[i * s + j] > out->buf[i] ? 
        in->buf[i * s + j] : out->buf[i];
    }
  }
}

/* Concat
 * @param [in1] in1: [N1]
 * @param [in2] in2: [N2]
 * @param [in3] in3: [N3]
 * @param [in4] in4: [N4]
 * @param [out] out: [N1 + N2 + N3 + N4]
 * 'N1', 'N2', 'N3', and 'N4' are the num of elems in the tensors.
 */
void Concat(Tensor *in1, Tensor *in2, Tensor *in3, Tensor *in4, 
            Tensor *out) {
  size_t N1 = in1->shape[0];
  size_t N2 = in2->shape[0];
  size_t N3 = in3->shape[0];
  size_t N4 = in4->shape[0];

  for (size_t i = 0; i < N1; i++) {
    out->buf[i] = in1->buf[i];
  }
  for (size_t i = 0; i < N2; i++) {
    out->buf[N1 + i] = in2->buf[i];
  }
  for (size_t i = 0; i < N3; i++) {
    out->buf[N1 + N2 + i] = in3->buf[i];
  }
  for (size_t i = 0; i < N4; i++) {
    out->buf[N1 + N2 + N3 + i] = in4->buf[i];
  }
}

/* Linear 
 * @param [in1]  in: [N]
 * @param [in2]   w: [M, N]
 * @param [in3]   b: [M]
 * @param [out] out: [M]
 * 'N' is the input feature size
 * 'M' is the output feature size
 */
void Linear(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
  size_t N = in->shape[0];
  size_t M = w->shape[0];

  for (size_t i = 0; i < M; i++) {
    float val = 0.f;
    for (size_t j = 0; j < N; j++) {
      val += in->buf[j] * w->buf[i * N + j];
    }
    out->buf[i] = val + b->buf[i];
  }
}
/* [Advanced Example] Linear in Half precision on CPU */
void Linear_Half(Tensor *in, Tensor *w, Tensor *b, Tensor *out) {
  size_t N = in->shape[0];
  size_t M = w->shape[0];

  for (size_t i = 0; i < M; i++) {
    float val = 0.f;
    for (size_t j = 0; j < N; j++) {
      val += static_cast<float>(half_cpu(in->buf[j]) * 
        half_cpu(w->buf[i * N + j]));
    }
    out->buf[i] = val + b->buf[i];
  }
}

/* Softmax (w/ Max Trick)
 * @param [in & out] inout: [N]
 * 'N' is the number of elements in the tensor.
 */
void Softmax(Tensor *inout) {
  size_t N = inout->shape[0];

  float max_val = -INFINITY;
  for (size_t i = 0; i < N; i++) {
    max_val = inout->buf[i] > max_val ? inout->buf[i] : max_val;
  }

  float sum = 0.f;
  for (size_t i = 0; i < N; i++) {
    inout->buf[i] = exp(inout->buf[i] - max_val);
    sum += inout->buf[i];
  }

  for (size_t i = 0; i < N; i++) { inout->buf[i] /= sum; }
}

/* (Elemwise) Scaling
 * @param [in & out] inout: [N]
 * @param [in]           s: [1]
 * 'N' is the number of elements in the tensor.
 */
void Scaling(Tensor *inout, float s) {
  size_t N = inout->shape[0];

  for (size_t i = 0; i < N; i++) {
    inout->buf[i] *= s;
  }
}

/* (Elemwise) Addition
 * @param [in1] in1: [N]
 * @param [in2] in2: [N]
 * @param [in3] in3: [N]
 * @param [in4] in4: [N]
 * @param [out] out: [N]
 * 'N' is the number of elements in the input tensor.
 */
void Add(Tensor *in1, Tensor *in2, Tensor *in3, Tensor *in4, 
         Tensor *out) {
  size_t N = in1->shape[0];

  for (size_t i = 0; i < N; i++) {
    out->buf[i] = in1->buf[i] + in2->buf[i] + in3->buf[i] + in4->buf[i];
  }
}

